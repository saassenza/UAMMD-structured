#include "hip/hip_runtime.h"
#include "System/ExtendedSystem.cuh"
#include "GlobalData/GlobalData.cuh"
#include "ParticleData/ExtendedParticleData.cuh"
#include "ParticleData/ParticleGroup.cuh"

#include "Interactor/Pair/PairInteractor.cuh"
#include "Interactor/Pair/NonBonded/NonBonded.cuh"
#include "Interactor/InteractorFactory.cuh"

#include "Interactor/BasicPotentials/DebyeHuckel.cuh"
#include "Interactor/BasicPotentials/LennardJones.cuh"
#include "Interactor/BasicParameters/Pair/WCA_DH.cuh"
#include "Utils/ParameterHandler/PairParameterHandler.cuh"

namespace uammd{
namespace structured{
namespace Potentials{
namespace NonBonded{

    struct softWCA_DH_{

	using ParametersType        = typename BasicParameters::Pairs::WCA_DH;
	using ParameterPairsHandler = typename structured::PairParameterHandler<ParametersType>;
        using ParametersPairsIterator = typename ParameterPairsHandler::PairIterator;

        //Computational data
        struct ComputationalData{

            real4* pos;

            Box    box;

            real ELECOEF;

            real dielectricConstant;
            real debyeLength;
	    real lambda;
            real alpha;
            int  n;

	    ParametersPairsIterator paramPairIterator;

            real cutOffFactor;
            real cutOff;
        };

        //Potential parameters
        struct StorageData{

	    std::shared_ptr<ParameterPairsHandler> Param;
            real ELECOEF;

            real dielectricConstant;
            real debyeLength;
            real alpha;
            int  n;

            real cutOffFactor;
            real cutOff;
        };

        static __host__ ComputationalData getComputationalData(std::shared_ptr<GlobalData>    gd,
                                                               std::shared_ptr<ParticleGroup> pg,
                                                               const StorageData&  storage,
                                                               const Computables& comp,
                                                               const hipStream_t& st){

            ComputationalData computational;

            std::shared_ptr<ParticleData> pd = pg->getParticleData();

            computational.pos    = pd->getPos(access::location::gpu, access::mode::read).raw();

            computational.box = gd->getEnsemble()->getBox();
	    computational.lambda = gd->getEnsemble()->getLambda();
	    computational.paramPairIterator = storage.Param->getPairIterator();

            computational.ELECOEF = storage.ELECOEF;

            computational.dielectricConstant = storage.dielectricConstant;
            computational.debyeLength = storage.debyeLength;
            computational.alpha = storage.alpha;
            computational.n = storage.n;

            computational.cutOffFactor = storage.cutOffFactor;
            computational.cutOff = storage.cutOff;

            return computational;
        }

        //Storage data reader

        static __host__ StorageData getStorageData(std::shared_ptr<GlobalData>    gd,
                                                   std::shared_ptr<ParticleGroup> pg,
                                                   DataEntry& data){

            StorageData storage;

            storage.ELECOEF = gd->getUnits()->getElectricConversionFactor();

            storage.dielectricConstant = data.getParameter<real>("dielectricConstant");
            storage.debyeLength        = data.getParameter<real>("debyeLength");
            storage.alpha              = data.getParameter<real>("alpha");
            storage.n                  = data.getParameter<real>("n");
	    storage.Param = std::make_shared<ParameterPairsHandler>(gd,pg,data);

            storage.cutOffFactor = data.getParameter<real>("cutOffFactor");
            storage.cutOff       = storage.cutOffFactor*storage.debyeLength;

	    auto pairsParam = storage.Param->getPairParameters();
	    real maxSigma = 0.0;
            for(auto p : pairsParam){
                maxSigma=std::max(maxSigma,p.second.sigma);
            }
	    if (maxSigma > storage.cutOff)
	    {
		    storage.cutOff = maxSigma;
	    }

            System::log<System::MESSAGE>("[softWCA_DH] cutOff: %f" ,storage.cutOff);

            return storage;

        }


        static inline __device__ real energy(const int index_i,const int index_j,
                                             const ComputationalData& computational){

            const real4 posi = computational.pos[index_i];
            const real4 posj = computational.pos[index_j];

            const real3 rij = computational.box.apply_pbc(make_real3(posj)-make_real3(posi));
            const real r2   = dot(rij, rij);

            real e = real(0.0);

            real cutOff2 = computational.cutOff*computational.cutOff;
            const real chgProduct = computational.paramPairIterator(index_i,index_j).chargeProduct;
            if(r2>0 and r2<=cutOff2 and chgProduct != real(0.0)){

                e+=BasicPotentials::DebyeHuckel::DebyeHuckel::energy(rij,r2,
                                                                     computational.ELECOEF,
                                                                     chgProduct,
                                                                     computational.dielectricConstant,
                                                                     computational.debyeLength);
            }

	    const real epsilon  = computational.paramPairIterator(index_i,index_j).epsilon;
            const real sigma    = computational.paramPairIterator(index_i,index_j).sigma;
            const real lambda   = computational.lambda;
            const real alpha    = computational.alpha;
            const int n         = computational.n;
	    const real Acomodo  = alpha*(real(1.0)-lambda)*(real(1.0)-lambda); 
	    const real minPos   = sigma*pow(real(1.0) - Acomodo, real(1.0)/real(6.0));
	    const real minPos2  = minPos*minPos;	    //position of minimum
	    if (r2 < minPos2)
	    {
		    e += BasicPotentials::LennardJones::SoftCoreType2::energy(rij,r2,epsilon,sigma,lambda,alpha,n);
		    const real3 rij0 = make_real3(0.0);
		    e += -BasicPotentials::LennardJones::SoftCoreType2::energy(rij0,minPos2,epsilon,sigma,lambda,alpha,n);
	    }

            return e;
        }

      static inline __device__ real3 force(const int index_i,const int index_j,
                                             const ComputationalData& computational){

            const real4 posi = computational.pos[index_i];
            const real4 posj = computational.pos[index_j];

            const real3 rij = computational.box.apply_pbc(make_real3(posj)-make_real3(posi));
            const real r2   = dot(rij, rij);

            real3 f = make_real3(real(0.0));

            real cutOff2 = computational.cutOff*computational.cutOff;
            const real chgProduct = computational.paramPairIterator(index_i,index_j).chargeProduct;
            if(r2>0 and r2<=cutOff2 and chgProduct != real(0.0)){

                f+=BasicPotentials::DebyeHuckel::DebyeHuckel::force(rij,r2,
                                                                    computational.ELECOEF,
                                                                    chgProduct,
                                                                    computational.dielectricConstant,
                                                                    computational.debyeLength);
            }

	    const real epsilon = computational.paramPairIterator(index_i,index_j).epsilon;
            const real sigma   = computational.paramPairIterator(index_i,index_j).sigma;
            const real lambda  = computational.lambda;
            const real alpha   = computational.alpha;
            const int n        = computational.n;
	    const real Acomodo = alpha*(real(1.0)-lambda)*(real(1.0)-lambda); 
	    const real minPos  = sigma*pow(real(1.0) - Acomodo, real(1.0)/real(6.0));
	    const real minPos2 = minPos*minPos;	    //position of minimum
	    if (r2 < minPos2)
            {
                    f += BasicPotentials::LennardJones::SoftCoreType2::force(rij,r2,epsilon,sigma,lambda,alpha,n);
            }

            return f;
        }

      static inline __device__ tensor3 hessian(const int index_i,const int index_j,
                                               const ComputationalData& computational){

            const real4 posi = computational.pos[index_i];
            const real4 posj = computational.pos[index_j];

            const real3 rij = computational.box.apply_pbc(make_real3(posj)-make_real3(posi));
            const real r2   = dot(rij, rij);

            tensor3 H = tensor3(real(0.0));

            real cutOff2 = computational.cutOff*computational.cutOff;
            const real chgProduct = computational.paramPairIterator(index_i,index_j).chargeProduct;
            if(r2>0 and r2<=cutOff2 and chgProduct != real(0.0)){

                H+=BasicPotentials::DebyeHuckel::DebyeHuckel::hessian(rij,r2,
                                                                      computational.ELECOEF,
                                                                      chgProduct,
                                                                      computational.dielectricConstant,
                                                                      computational.debyeLength);
            }

            const real epsilon = computational.paramPairIterator(index_i,index_j).epsilon;
            const real sigma   = computational.paramPairIterator(index_i,index_j).sigma;
            const real lambda  = computational.lambda;
            const real alpha   = computational.alpha;
            const int n        = computational.n;
	    const real Acomodo = sigma*alpha*(real(1.0)-lambda)*(real(1.0)-lambda); 
	    const real minPos       = sigma*pow(real(1.0) - Acomodo, real(1.0)/real(6.0));
	    const real minPos2      = minPos*minPos;	    //position of minimum
	    if (r2 < minPos2)
            {
                    H += BasicPotentials::LennardJones::SoftCoreType2::hessian(rij,r2,epsilon,sigma,lambda,alpha,n);
            }

            return H;
        }

      static inline __device__ real lambdaDerivative(int index_i, int index_j,
                                                       const ComputationalData& computational){

            const real4 posi = computational.pos[index_i];
            const real4 posj = computational.pos[index_j];

            const real3 rij = computational.box.apply_pbc(make_real3(posj)-make_real3(posi));

            const real epsilon = computational.paramPairIterator(index_i,index_j).epsilon;
            const real sigma   = computational.paramPairIterator(index_i,index_j).sigma;
            const real lambda  = computational.lambda;
            const real alpha   = computational.alpha;
            const int  n       = computational.n;

            const real r2 = dot(rij, rij);
	    const real Acomodo = sigma*alpha*(real(1.0)-lambda)*(real(1.0)-lambda); 
	    const real minPos       = sigma*pow(real(1.0) - Acomodo, real(1.0)/real(6.0));
	    const real minPos2      = minPos*minPos;	    //position of minimum

            real ld = real(0.0);

            if (r2 < minPos2)
	    {
                ld = BasicPotentials::LennardJones::SoftCoreType2::lambdaDerivative(rij,r2,epsilon,sigma,lambda,alpha,n);
            }

            return ld;

        }
    };

    using softWCA_DH = NonBondedHessian_<softWCA_DH_>;

}}}}

REGISTER_NONBONDED_INTERACTOR(
    NonBonded,softWCA_DH,
    uammd::structured::Interactor::PairInteractor<uammd::structured::Potentials::NonBonded::softWCA_DH>
)
